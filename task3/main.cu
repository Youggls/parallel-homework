#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "util.h"
#include <iostream>

//CPU对照组，用于对比加速比
void sumMatrix2DonCPU(float* MatA, float* MatB, float* MatC, int nx, int ny)
{
    float* a = MatA;
    float* b = MatB;
    float* c = MatC;
    for(int j = 0; j < ny; j++) {
        for(int i = 0; i < nx; i++) {
            c[i] = a[i] + b[i];
        }
        c += nx;
        b += nx;
        a += nx;
    }
}

void matrixMulCpu(float* fpMatrixA, float* fpMatrixB, float* fpMatrixC, int m, int n, int k) {
    float sum = 0.0f;
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            for(int l = 0; l < k; l++) {
                sum += fpMatrixA[i * k + l] * fpMatrixB[l * n + j];
            }
            fpMatrixC[i * n + j] = sum;
            sum = 0.0f;
        }
    }
}

void arrayAddMatrixCpu(float* fpMatrixA, float* fpMatrixB, float* fpMatrixC, int m, int n) {
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            fpMatrixC[i * n + j] = fpMatrixA[i * n + j] + fpMatrixB[j];
        }
    }
}

//核函数，每一个线程计算矩阵中的一个元素。
__global__ void sumMatrix(float* MatA, float* MatB, float* MatC, int nx, int ny) {
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = ix + iy * ny;
    if (ix<nx && iy<ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

__global__ void arrayAddMatrix(const float* a, const float* b, float* c, int m, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = i + j * n;
    if (i < m && j < n) {
        c[idx]= a[idx] + b[j];
    }
}

__global__ void matrixAddMatrix(const float* a, const float* b, float* c, int m, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = i + j * n;
    if (i < m && j < n) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void matrixMulGlobalKernel(float* pfMatrixA, float* pfMatrixB, float* pfMatrixC, int m, int n, int k) {
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    float fCVal = 0.0f;
    for(int i = 0; i < k; i++) {
        fCVal += pfMatrixA[nRow * k + i] * pfMatrixB[i * n + nCol];
    }
    pfMatrixC[nRow * n + nCol] = fCVal;
}

void test(int batchSize, int featureSize ,int hiddenSize, int outSize) {
    printf("%d,%d,%d,", batchSize, featureSize, hiddenSize);
    int weight1NBytes = featureSize * hiddenSize * sizeof(float);
    int weight2NBytes = hiddenSize * outSize * sizeof(float);
    float* weight1 = (float*)malloc(weight1NBytes);
    float* weight2 = (float*)malloc(weight2NBytes);
    float* bias1 = (float*)malloc(hiddenSize * sizeof(float));
    float* bias2 = (float*)malloc(outSize * sizeof(float));
    initialData(weight1, featureSize * hiddenSize);
    initialData(weight2, hiddenSize * outSize);
    initialData(bias1, hiddenSize);
    initialData(bias2, outSize);
    float* weight1Dev = nullptr;
    float* weight2Dev = nullptr;
    float* bias1Dev = nullptr;
    float* bias2Dev = nullptr;
    CHECK(hipMalloc((void**)&weight1Dev, weight1NBytes));
    CHECK(hipMalloc((void**)&weight2Dev, weight2NBytes));
    CHECK(hipMalloc((void**)&bias1Dev, hiddenSize * sizeof(float)));
    CHECK(hipMalloc((void**)&bias2Dev, outSize * sizeof(float)));
    CHECK(hipMemcpy(weight1Dev, weight1, weight1NBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(weight2Dev, weight2, weight2NBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(bias1Dev, bias1, hiddenSize * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(bias2Dev, bias2, outSize * sizeof(float), hipMemcpyHostToDevice));
    int batchDataNBytes = batchSize * featureSize * sizeof(float);
    int tempDataNBytes = batchSize * hiddenSize * sizeof(float);
    int resultDataNBytes = batchSize * outSize * sizeof(float);
    float* batchData = (float*)malloc(batchDataNBytes);
    float* tempData = (float*)malloc(tempDataNBytes);
    float* resultData = (float*)malloc(resultDataNBytes);
    initialData(batchData, batchSize * featureSize);
    initialData(tempData, batchSize * hiddenSize);
    initialData(resultData, batchSize * outSize);
    float* batchDataDev = nullptr;
    float* tempDataDev = nullptr;
    float* resultDataDev = nullptr;
    CHECK(hipMalloc((void**)&batchDataDev, batchDataNBytes));
    CHECK(hipMalloc((void**)&tempDataDev, tempDataNBytes));
    CHECK(hipMalloc((void**)&resultDataDev, resultDataNBytes));
    CHECK(hipMemcpy(batchDataDev, batchData, batchDataNBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(tempDataDev, tempData, tempDataNBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(resultDataDev, resultData, resultDataNBytes, hipMemcpyHostToDevice));
    dim3 block(32, 32);
    dim3 grid((batchSize - 1) / block.x + 1, (featureSize - 1) / block.y + 1);
    double gpuStart = cpuSecond();
    matrixMulGlobalKernel<<<grid, block>>>(batchDataDev, weight1Dev, tempDataDev, batchSize, featureSize, hiddenSize);
    arrayAddMatrix<<<grid, block>>>(tempDataDev, bias1Dev, tempDataDev, batchSize, hiddenSize);
    matrixMulGlobalKernel<<<grid, block>>>(tempDataDev, weight2Dev, resultDataDev, batchSize, hiddenSize, outSize);
    arrayAddMatrix<<<grid, block>>>(resultDataDev, bias2Dev, resultDataDev, batchSize, outSize);
    double gpuTime = cpuSecond() - gpuStart;
    hipFree(batchDataDev);
    hipFree(tempDataDev);
    hipFree(resultDataDev);
    free(batchData);
    free(tempData);
    free(resultData);
    hipFree(weight1Dev);
    hipFree(weight2Dev);
    hipFree(bias1Dev);
    hipFree(bias2Dev);
    free(weight1);
    free(weight2);
    free(bias1);
    free(bias2);
    printf("%f\n", gpuTime * 1000);
}

int main(int argc, char** argv) {
    int batchSize = 64;
    int featureSize = 1024;
    int hiddenSize = 1024;
    int outSize = 32;
    for (size_t i = 1; i < argc; i += 2) {
        if (strcmp(argv[i], "-d") == 0) {
            batchSize = atoi(argv[i + 1]);
        } else if (strcmp(argv[i], "-f") == 0) {
            featureSize = atoi(argv[i + 1]);
        } else if (strcmp(argv[i], "-h") == 0) {
            hiddenSize = atoi(argv[i + 1]);
        } else if (strcmp(argv[i], "-o") == 0) {
            outSize = atoi(argv[i + 1]);
        } else {
            std::cout << argv[i];
            std::cerr << " Invalid argument" << std::endl;
            return 1;
        }
    }
    // printf("strating...\n");
    initDevice(0);
    test(batchSize, featureSize, hiddenSize, outSize);
    // // 输入二维矩阵，4096 * 4096，单精度浮点型。
    // int batchSizeArr[] = {512, 1024, 2048};
    // int featureSizeArr[] = {256, 512, 1024};
    // int hiddenSizeArr[] = {2048, 4096, 8196};
    // for (int i = 0; i < 3; i++) {
    //     for (int j = 0; j < 3; j++) {
    //         for (int k = 0; k < 3; k++) {
    //             test(batchSizeArr[i], featureSizeArr[j], hiddenSizeArr[k], 32);
    //         }
    //     }
    // }
    // Malloc，开辟主机内存
    // float* A_host = (float*)malloc(nBytes);
    // float* B_host = (float*)malloc(nBytes);
    // float* C_host = (float*)malloc(nBytes);
    // float* C_from_gpu = (float*)malloc(nBytes);
    // initialData(A_host, nx * ny);
    // initialData(B_host, nx * ny);

    // // hipMalloc，开辟设备内存
    // float* A_dev = NULL;
    // float* B_dev = NULL;
    // float* C_dev = NULL;
    // CHECK(hipMalloc((void**)&A_dev, nBytes));
    // CHECK(hipMalloc((void**)&B_dev, nBytes));
    // CHECK(hipMalloc((void**)&C_dev, nBytes));

    // // 输入数据从主机内存拷贝到设备内存
    // CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
    // CHECK(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));

    // dim3 block(32, 32);
    // dim3 grid((nx - 1) / block.x + 1, (ny - 1) / block.y + 1);

    // // 测试 GPU 执行时间
    // double gpuStart = cpuSecond();
    // // 将核函数放在线程网格中执行
    // matrixAddMatrix<<<grid, block>>>(A_dev, B_dev, C_dev, nx, ny);
    // CHECK(hipDeviceSynchronize());
    // double gpuTime = cpuSecond() - gpuStart;
    // printf("GPU Execution Time: %f sec\n", gpuTime);

    // // 在 CPU 上完成相同的任务
    // hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost);
    // double cpuStart = cpuSecond();
    // sumMatrix2DonCPU(A_host, B_host, C_host, nx, ny);
    // double cpuTime = cpuSecond() - cpuStart;
    // printf("CPU Execution Time: %f sec\n", cpuTime);

    // // 检查 GPU 与 CPU 计算结果是否相同
    // CHECK(hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost));
    // checkResult(C_host, C_from_gpu, nx * ny);

    // // 释放内存
    // hipFree(A_dev);
    // hipFree(B_dev);
    // hipFree(C_dev);
    // free(A_host);
    // free(B_host);
    // free(C_host);
    // free(C_from_gpu);
    hipDeviceReset();
    return 0;
}